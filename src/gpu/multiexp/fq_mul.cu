// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// 
// We would like to acknowledge Niall Emmart at Nvidia for his significant
// contribution of concepts and code for generating efficient SASS on
// Nvidia GPUs. The following papers may be of interest:
//     Optimizing Modular Multiplication for NVIDIA's Maxwell GPUs
//     https://ieeexplore.ieee.org/document/7563271
//
//     Faster modular exponentiation using double precision floating point
//     arithmetic on the GPU
//     https://ieeexplore.ieee.org/document/8464792

#include <stdint.h>
#include <stdio.h>

#ifndef __FQ_MUL_CU__
#define __FQ_MUL_CU__

#define FQ_BITS 384
#define FQ_LIMBS (384/32)

__device__ __forceinline__ uint32_t add_cc(uint32_t a, uint32_t b) {
  uint32_t r;

  asm volatile ("add.cc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
  return r;
}

__device__ __forceinline__ uint32_t addc_cc(uint32_t a, uint32_t b) {
  uint32_t r;

  asm volatile ("addc.cc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
  return r;
}

__device__ __forceinline__ uint32_t addc(uint32_t a, uint32_t b) {
  uint32_t r;

  asm volatile ("addc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
  return r;
}


__device__ __forceinline__ uint32_t madlo(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("mad.lo.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

__device__ __forceinline__ uint32_t madlo_cc(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("mad.lo.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

__device__ __forceinline__ uint32_t madloc_cc(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("madc.lo.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

__device__ __forceinline__ uint32_t madloc(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("madc.lo.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

__device__ __forceinline__ uint32_t madhi(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("mad.hi.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

__device__ __forceinline__ uint32_t madhi_cc(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("mad.hi.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

__device__ __forceinline__ uint32_t madhic_cc(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("madc.hi.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

__device__ __forceinline__ uint32_t madhic(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("madc.hi.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

typedef struct {
  int32_t _position;
} chain_t;

__device__ __forceinline__
void chain_init(chain_t *c) {
  c->_position = 0;
}

__device__ __forceinline__
uint32_t chain_add(chain_t *ch, uint32_t a, uint32_t b) {
  uint32_t r;
  
  ch->_position++;
  if(ch->_position==1)
    r=add_cc(a, b);
  else
    r=addc_cc(a, b);
  return r;
}

__device__ __forceinline__
uint32_t chain_madlo(chain_t *ch, uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;
  
  ch->_position++;
  if(ch->_position==1)
    r=madlo_cc(a, b, c);
  else
    r=madloc_cc(a, b, c);
  return r;
}

__device__ __forceinline__
uint32_t chain_madhi(chain_t *ch, uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;
  
  ch->_position++;
  if(ch->_position==1)
    r=madhi_cc(a, b, c);
  else
    r=madhic_cc(a, b, c);
  return r;
}

// Requirement: yLimbs >= xLimbs
__device__ __forceinline__
void mult_v1(uint32_t *x, uint32_t *y, uint32_t *xy) {
  const uint32_t xLimbs  = FQ_LIMBS;
  const uint32_t yLimbs  = FQ_LIMBS;
  const uint32_t xyLimbs = xLimbs + yLimbs;
  uint32_t temp[xyLimbs];
  uint32_t carry = 0;

  #pragma unroll
  for (int32_t i = 0; i < xyLimbs; i++) {
    temp[i] = 0;
  }
  
  #pragma unroll
  for (int32_t i = 0; i < xLimbs; i++) {
    chain_t chain1;
    chain_init(&chain1);
    #pragma unroll
    for (int32_t j = 0; j < yLimbs; j++) {
      if ((i + j) % 2 == 1) {
        temp[i + j - 1] = chain_madlo(&chain1, x[i], y[j], temp[i + j - 1]);
        temp[i + j]     = chain_madhi(&chain1, x[i], y[j], temp[i + j]);
      }      
    }
    if (i % 2 == 1) {
      temp[i + yLimbs - 1] = chain_add(&chain1, 0, 0);
    }
  }

  #pragma unroll
  for (int32_t i = xyLimbs - 1; i > 0; i--) {
    temp[i] = temp[i - 1];
  }
  temp[0] = 0;
  
  #pragma unroll
  for (int32_t i = 0; i < xLimbs; i++) {
    chain_t chain2;
    chain_init(&chain2);
    
    #pragma unroll
    for (int32_t j = 0; j < yLimbs; j++) {
      if ((i + j) % 2 == 0) {
        temp[i + j]     = chain_madlo(&chain2, x[i], y[j], temp[i + j]);
        temp[i + j + 1] = chain_madhi(&chain2, x[i], y[j], temp[i + j + 1]);
      }  
    }
    if ((i + yLimbs) % 2 == 0 && i != yLimbs - 1) {
      temp[i + yLimbs]     = chain_add(&chain2, temp[i + yLimbs], carry);
      temp[i + yLimbs + 1] = chain_add(&chain2, temp[i + yLimbs + 1], 0);
      carry = chain_add(&chain2, 0, 0);
    }
    if ((i + yLimbs) % 2 == 1 && i != yLimbs - 1) {
      carry = chain_add(&chain2, carry, 0);
    }
  }

  #pragma unroll
  for(int32_t i = 0; i < xyLimbs; i++) {
    xy[i] = temp[i];
  }
}

__device__ void reduce(uint32_t accLow[FQ_LIMBS], uint32_t np0, uint32_t fq[FQ_LIMBS]) {
  // accLow is an IN and OUT vector
  // count must be even
  const uint32_t count = FQ_LIMBS;
  uint32_t accHigh[count];
  uint32_t bucket=0, lowCarry=0, highCarry=0, q;
  int32_t  i, j;

  #pragma unroll
  for(i=0;i<count;i++)
    accHigh[i]=0;

  // bucket is used so we don't have to push a carry all the way down the line

  #pragma unroll
  for(j=0;j<count;j++) {       // main iteration
    if(j%2==0) {
      add_cc(bucket, 0xFFFFFFFF);
      accLow[0]=addc_cc(accLow[0], accHigh[1]);
      bucket=addc(0, 0);

      q=accLow[0]*np0;

      //chain_t<0,false,true> chain1;
      chain_t chain1;
      chain_init(&chain1);

      #pragma unroll
      for(i=0;i<count;i+=2) {
        accLow[i]=chain_madlo(&chain1, q, fq[i], accLow[i]);
        accLow[i+1]=chain_madhi(&chain1, q, fq[i], accLow[i+1]);
      }
      lowCarry=chain_add(&chain1, 0, 0);

      chain_t chain2;
      chain_init(&chain2);
      for(i=0;i<count-2;i+=2) {
        accHigh[i]=chain_madlo(&chain2, q, fq[i+1], accHigh[i+2]);    // note the shift down
        accHigh[i+1]=chain_madhi(&chain2, q, fq[i+1], accHigh[i+3]);
      }
      accHigh[i]=chain_madlo(&chain2, q, fq[i+1], highCarry);
      accHigh[i+1]=chain_madhi(&chain2, q, fq[i+1], 0);
    }
    else {
      add_cc(bucket, 0xFFFFFFFF);
      accHigh[0]=addc_cc(accHigh[0], accLow[1]);
      bucket=addc(0, 0);

      q=accHigh[0]*np0;

      chain_t chain3;
      chain_init(&chain3);
      #pragma unroll
      for(i=0;i<count;i+=2) {
        accHigh[i]=chain_madlo(&chain3, q, fq[i], accHigh[i]);
        accHigh[i+1]=chain_madhi(&chain3, q, fq[i], accHigh[i+1]);
      }
      highCarry=chain_add(&chain3, 0, 0);

      chain_t chain4;
      chain_init(&chain4);
      for(i=0;i<count-2;i+=2) {
        accLow[i]=chain_madlo(&chain4, q, fq[i+1], accLow[i+2]);    // note the shift down
        accLow[i+1]=chain_madhi(&chain4, q, fq[i+1], accLow[i+3]);
      }
      accLow[i]=chain_madlo(&chain4, q, fq[i+1], lowCarry);
      accLow[i+1]=chain_madhi(&chain4, q, fq[i+1], 0);
    }
  }

  // at this point, accHigh needs to be shifted back a word and added to accLow
  // we'll use one other trick.  Bucket is either 0 or 1 at this point, so we
  // can just push it into the carry chain.

  chain_t chain5;
  chain_init(&chain5);
  chain_add(&chain5, bucket, 0xFFFFFFFF);    // push the carry into the chain
  #pragma unroll
  for(i=0;i<count-1;i++)
    accLow[i]=chain_add(&chain5, accLow[i], accHigh[i+1]);
  accLow[i]=chain_add(&chain5, accLow[i], highCarry);
}


// Nvidia only!
__noinline__
__device__ Fq Fq_mul(Fq a, Fq b) {
  // Perform full multiply
  Fq_limb ab[2 * Fq_LIMBS];
  mult_v1(a.val, b.val, ab);

  uint32_t io[Fq_LIMBS];
  #pragma unroll
  for(int i=0;i<Fq_LIMBS;i++) {
    io[i]=ab[i];
  }
  reduce(io, Fq_INV, Fq_P.val);

  // Add io to the upper words of ab
  ab[Fq_LIMBS] = add_cc(ab[Fq_LIMBS], io[0]);
  int j;
  #pragma unroll
  for (j = 1; j < Fq_LIMBS - 1; j++) {
    ab[j + Fq_LIMBS] = addc_cc(ab[j + Fq_LIMBS], io[j]);
  }
  ab[2 * Fq_LIMBS - 1] = addc(ab[2 * Fq_LIMBS - 1], io[Fq_LIMBS - 1]);

  Fq r;
  #pragma unroll
  for (int i = 0; i < Fq_LIMBS; i++) {
    r.val[i] = ab[i + Fq_LIMBS];
  }

  if (Fq_gte(r, Fq_P)) {
    r = Fq_sub_(r, Fq_P);
  }

  return r;
}

#endif  // __FQ_MUL_CU__
